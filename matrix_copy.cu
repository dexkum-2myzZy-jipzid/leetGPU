#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void copy_matrix_kernel(const float* A, float* B, int N) {
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for(int i = pos; i < N * N; i += stride){
        B[i] = A[i];
    }

}

// A, B are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, float* B, int N) {
    int total = N * N;
    int threadsPerBlock = 256;
    int blocksPerGrid = (total + threadsPerBlock - 1) / threadsPerBlock;
    copy_matrix_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, N);
    hipDeviceSynchronize();
} 