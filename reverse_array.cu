#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void reverse_array(float* input, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N/2){
        float tmp = input[i];
        input[i] = input[N-1-i];
        input[N-1-i] = tmp;
    }

}

// input is device pointer
void solve(float* input, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    reverse_array<<<blocksPerGrid, threadsPerBlock>>>(input, N);
    hipDeviceSynchronize();
}